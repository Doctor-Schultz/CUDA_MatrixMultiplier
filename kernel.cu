﻿
#include "hip/hip_runtime.h"


#include <stdio.h>

// cudaError_t addWithCuda(int *c, const int *a, const int *b, unsigned int size);

// Matrix multiplication is really just a bunch of dot products,
// so I think it is intuitive to design the kernel to be a dot product function
__global__ void dotProductKernel()
{

}

int main()
{
    /*
    These are what the default matrices look like:

    A:
    1 2 3
    4 5 6
    
    B:
    10 11
    20 21
    30 31
    
    AB = C
    */

    // The dimensions of the matrices, change these if you are 
    // multiplying your own matrices
    const int A_numRows = 2;
    const int A_numCols = 3;
    const int B_numRows = 3;
    const int B_numCols = 2;

    int *A, * B, * C; // Host matrices
    int *dev_A, *dev_B, *dev_C; // Corresponding device matrices

    // Allocating the hsot memory
    A = (int*)malloc(A_numRows * A_numCols * sizeof(int));
    B = (int*)malloc(B_numRows * B_numCols * sizeof(int));

    // The output matrix dimensions will be A_numRows x B_numCols
    C = (int*)malloc(A_numRows * B_numCols * sizeof(int));


    // Now allocate the corresponding device memory
    hipMalloc((void**)&dev_A, A_numRows * A_numCols * sizeof(int));
    hipMalloc((void**)&dev_B, B_numRows * B_numCols * sizeof(int));
    hipMalloc((void**)&dev_C, A_numRows * B_numCols * sizeof(int));

    // This next part is just populating the matrices A and B



    return 0;
}